// CUDA Sample

// * header
#include <stdio.h>

#include "hip/hip_runtime.h"


// comment samples:
// ! hello world
// * another comment
// ? and another

/* 
 ! block comments are the same 
*/

// * kernel
__global__ 
void helloFromGPU (void) 
{
    printf("Hello World from GPU!\n");
}

// * main
int main(int argc, char* argv[])
{
    // hello from cpu
    printf("Hello World from CPU!\n");

    // hello from gpu
    for(int i=0;i<1000;i++)
    {
        helloFromGPU <<<1, 100>>>();
        hipDeviceReset();
    }
    
    return 0;
}
